#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <iostream>
#include <cstring>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

constexpr uint32_t WINDOW_WIDTH = 1280;
constexpr uint32_t WINDOW_HEIGHT = 720;
constexpr const char* WINDOW_TITLE = "CUDA-OpenGL Interop Test";

// RANDOM IMAGE
constexpr uint32_t IMAGE_WIDTH = 1280;
constexpr uint32_t IMAGE_HEIGHT = 720;

const char* vertexShaderSource = 
    "#version 330 core\n"
    "layout (location = 0) in vec3 aPos;\n"
    "layout (location = 1) in vec2 aTexCoord;\n"
    "void main() {\n"
    "   gl_Position = vec4(aPos, 1);\n"
    "   TexCoord = aTexCoord;\n"
    "}\0";

const char* fragmentShaderSource =
    "#version 330 core\n"
    "out vec4 fragColor;"
    "in vec2 TexCoord;\n"
    "uniform sampler2D texture1;\n"
    "void main() {\n"
    "   fragColor = texture(texture1, TexCoord);\n"
    "}\0";


enum Mode {
    CPU,
    CUDA,
    CUDA_OPENGL
};


int main(int argc, char* argv[]){

    if(argc != 2) {
        printf("Usage : mode <--cpu, --cuda, --cuda-opengl>\n");
        return EXIT_FAILURE;
    }
    
    Mode mode;
    if(strcmp(argv[1], "--cpu") == 0) {
        mode = CPU;
    } else if (strcmp(argv[1], "--cuda") == 0) {
        mode = CUDA;
    } else if (strcmp(argv[1], "--cuda-opengl") == 0) {
        mode = CUDA_OPENGL;
    }

    if(!glfwInit()){
        printf("Failed to initialize glfw\n");
        return EXIT_FAILURE;
    }

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    GLFWwindow* window = glfwCreateWindow(WINDOW_WIDTH, WINDOW_HEIGHT, WINDOW_TITLE, nullptr, nullptr);
    if(!window) {
        printf("Failed to create window\n");
        glfwTerminate();
        return EXIT_FAILURE;
    }
    glfwMakeContextCurrent(window);

    if(!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
        printf("Failed to initialize glad\n");
        glfwTerminate();
        return EXIT_FAILURE;
    }
    auto glVersion = glGetString(GL_VERSION);
    printf("OPENGL CONTEXT VERSION : %s\n", glVersion);

    uint32_t vertexShader = glCreateShader(GL_VERTEX_SHADER);
    uint32_t fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);

    glShaderSource(vertexShader, 1, &vertexShaderSource, nullptr);
    glCompileShader(vertexShader);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, nullptr);
    glCompileShader(fragmentShader);

    uint32_t shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);

    float vertices[] = {
        -0.5f, -0.5f, 0.0f, 0.0f, 0.0f,
         0.5f, -0.5f, 0.0f, 1.0f, 0.0f,
         0.0f,  0.5f, 0.0f, 0.5f, 1.0f
    };

    uint32_t vao, vbo;
    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);

    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), 0);
    glEnableVertexAttribArray(1);
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));

    uint32_t texture;
    glGenTextures(1, &texture);
    glBindTexture(GL_TEXTURE_2D, texture);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, IMAGE_WIDTH, IMAGE_HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);

    uint8_t* imageCPU;
    uchar4* imageCUDA;
    hipGraphicsResource_t cudaResource;
    uint32_t pbo;
    
    if(mode == Mode::CPU) {
        imageCPU = new uint8_t[IMAGE_WIDTH * IMAGE_HEIGHT * 4];
    } else if (mode == Mode::CUDA) {

    } else if 









    while(!glfwWindowShouldClose(window)){
        glfwPollEvents();
        glClearColor(0.1f, 0.2f, 0.3f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);
        glfwSwapBuffers(window);
    }

    if(mode == )


    glfwTerminate();

    return EXIT_SUCCESS;

}