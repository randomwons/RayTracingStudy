#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "glad/glad.h"
#include "GLFW/glfw3.h"
#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"
#include "glm/gtc/type_ptr.hpp"

#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>

#define PI 3.141592653589793f

constexpr GLuint WINDOW_WIDTH = 1920;
constexpr GLuint WINDOW_HEIGHT = 1080;
constexpr const char* WINDOW_TITLE = "Ray Tracing Study";

struct Ray {
    float3 origin, dir;
};

__device__ float3 normalize(float3 v){
    float invLen = rsqrt(v.x * v.x + v.y * v.y + v.z * v.z);
    return make_float3(v.x * invLen, v.y * invLen, v.z * invLen);
}

__device__ float3 color(const Ray& r) {
    float t = 0.5f*(r.dir.y + 1.0f);
    return make_float3(1.0f-t + 0.5 * t, 1.0f-t + 0.7 * t, 1.0f-t + 1.0 * t);
}

__global__ void generateRays(Ray* rays, float vFov, int width, int height) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x < width && y < height){
        int pid = y * width + x;

        Ray ray;
        ray.origin = {0, 0, 0};

        float aspectRatio = (float)width / (float)height;
        float vFovRad = vFov * (PI / 180);
        float vh = 2 * tanf(vFovRad / 2);
        float vw = vh * aspectRatio;

        float dirX = (float)x / (float)width * vw - 0.5 * vw;
        float dirY = (float)y / (float)height * vh - 0.5 * vh;
        ray.dir = {dirX, dirY, -1};
        
        rays[pid] = ray;
    }

}

__global__ void traceRays(Ray* rays, float3* frame, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x < width && y < height) {
        int pid = y * width + x;
        frame[pid] = color(rays[pid]);
    }
}


__global__ void FrameKernel(uchar3* result, float3* frame, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x < width && y < height){
        int pid = y * width + x;
        result[pid] = {
            static_cast<unsigned char>(__saturatef(frame[pid].x) * 255.0f),
            static_cast<unsigned char>(__saturatef(frame[pid].y) * 255.0f),
            static_cast<unsigned char>(__saturatef(frame[pid].z) * 255.0f)
        };
    }
}

int main(){

    if(!glfwInit()){
        const char* desc = nullptr;
        glfwGetError(&desc);
        printf("Faeild to initialize glfw : %s\n", desc);
        return EXIT_FAILURE;
    }
    
    auto window = glfwCreateWindow(WINDOW_WIDTH, WINDOW_HEIGHT, WINDOW_TITLE, nullptr, nullptr);
    if(!window) {
        printf("Failed to create glfw window\n");
        glfwTerminate();
        return EXIT_FAILURE;
    }
    glfwMakeContextCurrent(window);
    glfwSwapInterval(0);

    if(!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)){
        printf("Failed to initialize glad\n");
        glfwTerminate();
        return EXIT_FAILURE;
    }
    auto glVersion = glGetString(GL_VERSION);
    printf("OpenGL Context Version : %s\n", reinterpret_cast<const char*>(glVersion));

    glViewport(0, 0, WINDOW_WIDTH, WINDOW_HEIGHT);
    glDisable(GL_DEPTH_TEST);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0);


    // uint32_t threadLayoutX = 2;
    // uint32_t threadLayoutY = 16;
    // uint32_t blockLayoutX = (WINDOW_WIDTH - threadLayoutX + 1) / threadLayoutX;
    // uint32_t blockLayoutY = (WINDOW_HEIGHT - threadLayoutY + 1) / threadLayoutY;

    uint32_t threadLayoutX = 8;
    uint32_t threadLayoutY = 8;
    uint32_t blockLayoutX = WINDOW_WIDTH / threadLayoutX + 1;
    uint32_t blockLayoutY = WINDOW_HEIGHT / threadLayoutY + 1;
    dim3 threadLayout = dim3(threadLayoutX, threadLayoutY);
    dim3 blockLayout = dim3(blockLayoutX, blockLayoutY);

    GLuint texture;
    GLuint pbo;
    cudaGLSetGLDevice(0);
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &texture);
    glBindTexture(GL_TEXTURE_2D, texture);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, WINDOW_WIDTH, WINDOW_HEIGHT, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    size_t size = WINDOW_WIDTH * WINDOW_HEIGHT * 3;
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, size * sizeof(GLubyte), NULL, GL_DYNAMIC_COPY);
    cudaGLRegisterBufferObject(pbo);


    uchar3* d_pixelBuffer;
    hipMalloc(&d_pixelBuffer, sizeof(uchar3) * WINDOW_WIDTH * WINDOW_HEIGHT);

    Ray* d_rays;
    hipMalloc(&d_rays, sizeof(Ray) * WINDOW_WIDTH * WINDOW_HEIGHT);

    float3* d_frame;
    hipMalloc(&d_frame, sizeof(float3) * WINDOW_WIDTH * WINDOW_HEIGHT);

    float vFov = 90;

    while(!glfwWindowShouldClose(window)) {
        
        generateRays<<<blockLayout, threadLayout>>>(d_rays, vFov, WINDOW_WIDTH, WINDOW_HEIGHT);
        traceRays<<<blockLayout, threadLayout>>>(d_rays, d_frame, WINDOW_WIDTH, WINDOW_HEIGHT);
        FrameKernel<<<blockLayout, threadLayout>>>(d_pixelBuffer, d_frame, WINDOW_WIDTH, WINDOW_HEIGHT);
    
        void *d_ptr = nullptr;
        cudaGLMapBufferObject((void**)&d_ptr, pbo);
        hipMemcpy2D(d_ptr, WINDOW_WIDTH * 3, (void*)d_pixelBuffer, WINDOW_WIDTH * 3, WINDOW_WIDTH * 3, WINDOW_HEIGHT, hipMemcpyDeviceToDevice);
        cudaGLUnmapBufferObject(pbo);

        glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
        glBindTexture(GL_TEXTURE_2D, texture);
        glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, WINDOW_WIDTH, WINDOW_HEIGHT, GL_RGB, GL_UNSIGNED_BYTE, nullptr);

        glBegin(GL_QUADS);
        glTexCoord2f(0.0f, 1.0f); glVertex2f(0.0f, 0.0f);
        glTexCoord2f(0.0f, 0.0f); glVertex2f(0.0f, 1.0f);
        glTexCoord2f(1.0f, 0.0f); glVertex2f(1.0f, 1.0f);
        glTexCoord2f(1.0f, 1.0f); glVertex2f(1.0f, 0.0f);
        glEnd();

        glfwPollEvents();
        glfwSwapBuffers(window);
    }

    hipFree(d_frame);
    hipFree(d_rays);
    hipFree(d_pixelBuffer);
    
    //

    glfwTerminate();

    return EXIT_SUCCESS;

}